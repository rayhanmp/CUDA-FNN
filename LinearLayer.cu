#include "hip/hip_runtime.h"
// linear-layer.cu

// Compute y = W * x

#include <iostream>
#include <hip/hip_runtime.h>
#include "LinearLayer.cuh"

LinearLayer::LinearLayer(int in_dim, int out_dim) : in_dim(in_dim), out_dim(out_dim) {
    hipMalloc(&W_d, in_dim * out_dim * sizeof(float));
    hipMalloc(&b_d, out_dim * sizeof(float));

    // Initialize the weight and bias matrices with dummy values
    float* W_h = new float[in_dim * out_dim];
    float* b_h = new float[out_dim];

    std::srand(static_cast<unsigned int>(time(0)));
    for (int i = 0; i < in_dim * out_dim; ++i) {
        W_h[i] = (std::rand() % 100) / 100.0f;
    }   

    for (int i = 0; i < out_dim; ++i) {
        b_h[i] = 0.0f;
    }

    hipMemcpy(W_d, W_h, in_dim * out_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, out_dim * sizeof(float), hipMemcpyHostToDevice);

    delete[] W_h;
    delete[] b_h;
}

LinearLayer::~LinearLayer() {
    hipFree(W_d);
    hipFree(b_d);
}

void LinearLayer::forward(const float* x_d, float* y_d) {
    int threads = 256;
    int blocks = (out_dim + threads - 1) / threads;
    matvec_bias<<<blocks, threads>>>(W_d, x_d, b_d, y_d, out_dim, in_dim);
    hipDeviceSynchronize();
}

__global__ void matvec_bias(float* W, float* x, float* b, float* y, int M, int K) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M) {
        float sum = 0.0f;
        for (int j = 0; j < K; ++j) {
            sum += W[row * K + j] * x[j]; // Dot product of W[row]
        }
        y[row] = sum + b[row];
    }
}

