// test.cu

#include <hip/hip_runtime.h>
#include <iostream>
__global__ void hello() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    hello<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}
