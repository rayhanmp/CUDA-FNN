#include <iostream>
#include <hip/hip_runtime.h>
#include "LinearLayer.cuh"
#include "relu.cuh"

int main() {
    const int in_dim = 4;
    const int out_dim = 3;

    float x_h[in_dim] = {1, 2, 3, 4};
    float y_h[out_dim];

    float* x_d, *y_d;
    hipMalloc(&x_d, in_dim * sizeof(float));
    hipMalloc(&y_d, out_dim * sizeof(float));
    hipMemcpy(x_d, x_h, in_dim * sizeof(float), hipMemcpyHostToDevice);

    LinearLayer layer(in_dim, out_dim);
    layer.forward(x_d, y_d);

    hipMemcpy(y_h, y_d, out_dim * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Output:\n";
    for (int i = 0; i < out_dim; ++i)
        std::cout << "y[" << i << "] = " << y_h[i] << '\n';

    hipFree(x_d);
    hipFree(y_d);
    return 0;
}